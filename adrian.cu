#include "hip/hip_runtime.h"
/* This is a CUDA implementation of Jermann Quadrini 2013 AER  
 * We simulate a equilibrium that collateral constraint is always binding to check the accuracy of 
 * their linearization approach. Hopeully we can ind something that they missed. A main suspect 
 * is the asymmetry of policy functions.
 */

/* Includes, system */
#include <iostream>
#include <iomanip>
#include <fstream>
#include <string>

// Includes, Thrust
#include <thrust/functional.h>
#include <thrust/for_each.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
#include <thrust/tuple.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>

/* Includes, cuda */
// #include <hip/hip_runtime.h>
#include <hipblas.h>
// #include <helper_functions.h>
#include "cuda_helpers.h"

// Includes, C++ codes
#include "cppcode.h"

using namespace std;
using namespace thrust;

// Define an class that contains parameters and steady states
struct para_struct {
	// Accuracy controls
	int nk ;
	int nb ;
	int nz ;
	int nxxi;
	int nm1;
	int maxiter;
	double tol;
	double kwidth ;
	double bwidth ;
	double mkwidth;

	// Model parameters
	double aalpha;
	double bbeta ;
	double ddelta;
	double ttheta;
	double kkappa;
	double ttau  ;
	double xxibar;
	double zbar  ;
	double dbar  ;
	double rrhozz;
	double rrhozxxi;
	double rrhoxxiz;
	double rrhoxxixxi;
	double var_epsz;
	double var_epsxxi;
	double A[4];
	double Ssigma_e[4];

	// Steady States
	double kss;
	double nss;
	double css;
	double wss;
	double dss;
	double mmuss;
	double mkss;
	double yss;


	// Find steady state and find aalpha based steady state target
	__host__ __device__
	void complete() {
		// Fill A and Ssigma_e
		A[0] = rrhozz; A[2] = rrhozxxi;
		A[1] = rrhoxxiz; A[3] = rrhoxxixxi;
		Ssigma_e[0] = var_epsz*var_epsz;
		Ssigma_e[3] = var_epsxxi*var_epsxxi;

		// Find aalpha based on SS computation
		double kovern = pow(xxibar,1/(ttheta-1));
		double covern = pow(kovern,ttheta) - ddelta*kovern;
		mmuss = 1 - ( bbeta*(1-ddelta)-1+xxibar )/( xxibar*(1-bbeta*ttheta)  );
		aalpha = double(0.7/0.3)*(1/covern)*(1-mmuss)*(1-ttheta)*pow(kovern,ttheta);
		double G = ( (1-mmuss)*(1-ttheta)*pow(kovern,ttheta) ) / ( aalpha*covern );
		nss = G/(1+G);
		css = nss*covern;
		kss = nss*kovern;
		wss = aalpha*css/(1-nss);
		dss = css - wss*nss;
		mkss = (1-ddelta+(1-mmuss)*zbar*ttheta*pow(kss,ttheta-1)*pow(nss,1-ttheta))/css;
		yss = zbar*pow(kss,ttheta)*pow(nss,1-ttheta);
	};

	// Export parameters to a .m file in MATLAB syntax
	__host__
	void exportmatlab(std::string filename) {
		std::ofstream fileout(filename.c_str(), std::ofstream::trunc);
		// Accuracy Controls
		fileout << setprecision(16) << "nk=" << nk << ";"<< endl;
		fileout << setprecision(16) << "nb=" << nb << ";"<< endl;
		fileout << setprecision(16) << "nz=" << nz << ";"<< endl;
		fileout << setprecision(16) << "nxxi=" << nxxi << ";"<< endl;
		fileout << setprecision(16) << "nm1=" << nm1 << ";"<< endl;

		// Model Parameters
		fileout << setprecision(16) << "aalpha=" << aalpha << ";"<< endl;
		fileout << setprecision(16) << "bbeta=" << bbeta << ";"<< endl;
		fileout << setprecision(16) << "ddelta=" << ddelta << ";"<< endl;
		fileout << setprecision(16) << "ttheta=" << ttheta << ";"<< endl;
		fileout << setprecision(16) << "xxibar=" << xxibar << ";"<< endl;
		fileout << setprecision(16) << "zbar=" << zbar << ";"<< endl;
		fileout << setprecision(16) << "rrhozz=" << rrhozz << ";"<< endl;
		fileout << setprecision(16) << "rrhozxxi=" << rrhozxxi << ";"<< endl;
		fileout << setprecision(16) << "rrhoxxiz=" << rrhoxxiz << ";"<< endl;
		fileout << setprecision(16) << "rrhoxxixxi=" << rrhoxxixxi << ";"<< endl;
		fileout << setprecision(16) << "ssigmaepsz=" << sqrt(var_epsz) << ";"<< endl;
		fileout << setprecision(16) << "ssigmaepsxxi=" << sqrt(var_epsxxi) << ";"<< endl;

		// Steady States
		fileout << setprecision(16) << "kss=" << kss << ";"<< endl;
		fileout << setprecision(16) << "nss=" << nss << ";"<< endl;
		fileout << setprecision(16) << "css=" << css << ";"<< endl;
		fileout << setprecision(16) << "wss=" << wss << ";"<< endl;
		fileout << setprecision(16) << "dss=" << dss << ";"<< endl;
		fileout << setprecision(16) << "mmuss=" << mmuss << ";"<< endl;
		fileout << setprecision(16) << "mkss=" << mkss << ";"<< endl;
		fileout << setprecision(16) << "yss=" << yss << ";"<< endl;
		fileout.close();
	};
};

void guess_linear(const host_vector<double> K, const host_vector<double> Z, const host_vector<double> XXI, host_vector<double> & V1_low, host_vector<double> & V1_high, para_struct para, double factor_low, double factor_high) {
	// Initialize matrices
	int n = 9; int n_jump = 8; int n_shock = 2;
	host_vector<double> A(n*n,0); 
	host_vector<double> B(n*n,0); 
	host_vector<double> C(n*n_shock,0); 
	host_vector<double> rrho(n_shock*n_shock,0);
   	host_vector<double> Pphi(n*(n-n_jump+n_shock),0);

	// Fill in matrices.
	// HH Budget. Correct.
	B[0+3*n] = para.nss;
	B[0+2*n] = para.wss;
	B[0+4*n] = 1;
	B[0+1*n] = -1;

	// Labor Demand. Correct
	B[1+5*n] = (para.ttheta-1)*para.yss/para.nss;
	B[1+6*n] = (1-para.ttheta)*(1-para.mmuss)/para.nss;
	B[1+2*n] = -(1-para.ttheta)*(1-para.mmuss)*para.yss/(para.nss*para.nss);
	B[1+3*n] = -1;

	// Labor Supply. Correct
	B[2+1*n] = para.aalpha/(1-para.nss);
	B[2+2*n] = para.aalpha*para.css/((1-para.nss)*(1-para.nss));
	B[2+3*n] = -1;

	// Capital Demand. Correct.
	A[3+8*n] = para.bbeta; 
	B[3+1*n] = -(1-para.mmuss*para.xxibar)/(para.css*para.css); 
	B[3+5*n] = -para.xxibar/para.css; 
	C[3+1*n] = -para.mmuss*para.xxibar/para.css;

	// Resource Constraint. Correct
	A[4+0*n] = 1; 
	B[4+0*n] = 1-para.ddelta; 
	B[4+6*n] = 1; 
	B[4+1*n] = -1;

	// Financial Constraint. Fixed.
	A[5+0*n] = para.xxibar;
	B[5+6*n] = 1;
	C[5+1*n] = -para.xxibar*para.kss;

	// Output Definition. Correct
	C[6+0*n] = para.yss;
	B[6+0*n] = para.ttheta*para.yss/para.kss;
	B[6+2*n] = (1-para.ttheta)*para.yss/para.nss;
	B[6+6*n] = -1;

	// Investment Definition. Correct
	A[7+0*n] = 1;
	B[7+7*n] = 1;
	B[7+0*n] = 1-para.ddelta;

	// MK defintion:
	B[8+1*n] = -pow(para.css,-2)*(1-para.ddelta+(1-para.mmuss)*para.ttheta*para.yss/para.kss); 
	B[8+5*n] = -para.ttheta*para.yss/(para.css*para.kss); 
	B[8+6*n] = (1-para.mmuss)*para.ttheta/(para.css*para.kss); 
	B[8+0*n] = -(1-para.mmuss)*para.ttheta*para.yss*pow(para.kss,-2)/para.css;
	B[8+8*n] = -1;

	for (int i=0; i< n_shock*n_shock; i++) {
		rrho[i] = para.A[i];
	};

	// Call linear solver
	linearQZ(A.data(),B.data(),C.data(),rrho.data(),n,n_jump,n_shock,Pphi.data());

	// Create guesses.
	for (int i_k=0; i_k<para.nk; i_k++) {
		for (int i_z = 0; i_z < para.nz; i_z++) {
			for (int i_xxi = 0; i_xxi < para.nxxi; i_xxi++) {
				double temp = para.mkss+Pphi[8+0*9]*(K[i_k]-para.kss) + Pphi[8+1*9]*(log(Z[i_z])-log(para.zbar))+ Pphi[8+2*9]*(log(XXI[i_xxi])-log(para.xxibar));
				V1_low[i_k+para.nk*i_z+para.nk*para.nz*i_xxi] = factor_low*temp;
				V1_high[i_k+para.nk*i_z+para.nk*para.nz*i_xxi] = factor_high*temp;
			};
		};
	};

};

struct case1_hour {
	// Data Member are const coefficents and some model parameters
	double c0, c1, c_oneminusttheta, c_twominusttheta;
	para_struct para;

	// Construct a function of hour based on state and control variables 
	__host__ __device__
	case1_hour(double k, double z, double xxi, double m1, double zkttheta, para_struct _para) {
		c0 = (1-para.ddelta)*k*m1 - 1 + para.ddelta;
		c1 = (1-para.ddelta)*(1-k*m1+para.aalpha*para.ttheta/(1-para.ttheta));
		c_oneminusttheta = (1-1/xxi)*m1*zkttheta;
		c_twominusttheta = (1/xxi-1)*zkttheta*(m1+para.aalpha*para.ttheta/(k*(1-para.ttheta)));
		para = _para;
	};

	// The function of hour
	__host__ __device__
	double operator()(double n) {
		return c0 + c1*n + c_oneminusttheta*pow(n,1-para.ttheta) + c_twominusttheta*pow(n,2-para.ttheta);
	};

	__host__ __device__
	// The derivative of function
	double prime(double n) {
		return c1 + (1-para.ttheta)*c_oneminusttheta*pow(n,-para.ttheta) + (2-para.ttheta)*c_twominusttheta*pow(n,1-para.ttheta);
	};
};

struct case2_hour {
	// Data Member are const coefficents and some model parameters
	double c0,  c_oneminusttheta, c_minusttheta;
	para_struct para;

	// Construct a function of hour based on state and control variables 
	__host__ __device__
	case2_hour(double k, double z, double xxi, double m1, double zkttheta, para_struct _para) {
		double ddelta = para.ddelta;
		double ttheta = para.ttheta;
		double aalpha = para.aalpha;
		c0 = (1-ddelta)/m1;
		c_oneminusttheta = (ttheta*zkttheta/(m1*k)+(1-ttheta)*zkttheta/aalpha);
		c_minusttheta = -(1-ttheta)*zkttheta/aalpha;
		para = _para;
	};

	// The function of hour
	__host__ __device__
	double operator()(double n) {
		return c0 + c_oneminusttheta*pow(n,1-para.ttheta) + c_minusttheta*pow(n,-para.ttheta);
	};

	__host__ __device__
	// The derivative of function
	double prime(double n) {
		return c0 + (1-para.ttheta)*c_oneminusttheta*pow(n,-para.ttheta) + (-para.ttheta)*c_minusttheta*pow(n,-para.ttheta-1);
	};
};

// Eureka function check whether a tuple (STATE,SHOCK,SHADOW) can survive to next iteration
__host__ __device__
bool eureka(double k, double z, double xxi,
            double m1, double zkttheta, int i_z, int i_xi,
            double* K, 
			double* EM1_low, double* EM1_high, 
			para_struct para) {
	
	// Declare control variables
	double n, Y, MPK, kplus, c, mmu, w, lhs1;
	int i_kplus;
	double interp_low, interp_high;

	// Case 1: Binding
	n = newton(case1_hour(k,z,xxi,m1,zkttheta,para),0.0,1.0,0.3);
	// printf("Hour solved here is: %f\n",n);
	Y = zkttheta*pow(n,1-para.ttheta);
	MPK = para.ttheta*Y/k;
	kplus = Y/xxi;
	c = Y+(1-para.ddelta)*k-kplus;
	mmu = 1-(m1*c-1+para.ddelta)/MPK;	
	w = (1-mmu)*(1-para.ttheta)*Y/n;
	// d = c-w*n;
	i_kplus = fit2grid(kplus,para.nk,K);
	lhs1 = (1-xxi*mmu)/c;
	// interp_low = EM1_low[i_kplus+para.nk*(i_z+i_xi*para.nz)] + (kplus-K[i_kplus])*(EM1_low[i_kplus+1+para.nk*(i_z+i_xi*para.nz)]-EM1_low[i_kplus+para.nk*(i_z+i_xi*para.nz)])/(K[i_kplus+1]-K[i_kplus]);
	// interp_high = EM1_high[i_kplus+para.nk*(i_z+i_xi*para.nz)] + (kplus-K[i_kplus])*(EM1_high[i_kplus+1+para.nk*(i_z+i_xi*para.nz)]-EM1_high[i_kplus+para.nk*(i_z+i_xi*para.nz)])/(K[i_kplus+1]-K[i_kplus]);
	interp_low = EM1_low[i_kplus+i_z*para.nk+i_xi*para.nz*para.nk];
	interp_high = EM1_high[i_kplus+i_z*para.nk+i_xi*para.nz*para.nk];
	
	if (
		(para.bbeta*interp_low <= lhs1) &&
		(lhs1 <=para.bbeta*interp_high) &&
		(c>0) && (mmu>=0) && (w>=0) && (n>0) && (n<1)//  &&  
	   )
	{
		return true;
	};

	// Case 2: Not Binding
	n = newton(case2_hour(k,z,xxi,m1,zkttheta,para),0.0,1.0,0.3);
	Y = zkttheta*pow(n,1-para.ttheta);
	MPK = para.ttheta*Y/k;
	mmu = 0;
	c = (1-para.ddelta+MPK)/m1;	
	kplus = (1-para.ddelta)*k + Y - c;
	w = (1-para.ttheta)*Y/n;
	lhs1 = (1-xxi*mmu)/c;
	i_kplus = fit2grid(kplus,para.nk,K);
	// interp_low = EM1_low[i_kplus+para.nk*(i_z+i_xi*para.nz)] + (kplus-K[i_kplus])*(EM1_low[i_kplus+1+para.nk*(i_z+i_xi*para.nz)]-EM1_low[i_kplus+para.nk*(i_z+i_xi*para.nz)])/(K[i_kplus+1]-K[i_kplus]);
	// interp_high = EM1_high[i_kplus+para.nk*(i_z+i_xi*para.nz)] + (kplus-K[i_kplus])*(EM1_high[i_kplus+1+para.nk*(i_z+i_xi*para.nz)]-EM1_high[i_kplus+para.nk*(i_z+i_xi*para.nz)])/(K[i_kplus+1]-K[i_kplus]);
	interp_low = EM1_low[i_kplus+i_z*para.nk+i_xi*para.nz*para.nk];
	interp_high = EM1_high[i_kplus+i_z*para.nk+i_xi*para.nz*para.nk];
	if (
		(para.bbeta*interp_low <= lhs1) &&
		(lhs1 <=para.bbeta*interp_high) &&
		(c>0) && (w>=0) && (xxi*kplus>Y) && (n>0) && (n<1)
	   )
	{
		return true;
	} else {
		return false;
	};
};

// This functor yields RHS for each (k', k, z). Follwing examples in Thrust doc
struct shrink 
{
	// Data member
	double *K, *Z, *XXI;
	double *V1_low;
	double *V1_high;
	double *Vplus1_low;
	double *Vplus1_high;
	double *EM1_low;
	double *EM1_high;
	double *flag;
	para_struct para;

	// Construct this object, create util from _util, etc.
	__host__ __device__
	shrink(double* K_ptr, double* Z_ptr, double* XXI_ptr,
	double* V1_low_ptr,
	double* V1_high_ptr,
	double* Vplus1_low_ptr,
	double* Vplus1_high_ptr,
	double* EM1_low_ptr,
	double* EM1_high_ptr,
	double* flag_ptr,
	para_struct _para)
	{
		K = K_ptr; Z = Z_ptr; XXI = XXI_ptr;
		V1_low = V1_low_ptr;
		V1_high = V1_high_ptr;
		Vplus1_low = Vplus1_low_ptr;
		Vplus1_high = Vplus1_high_ptr;
		EM1_low = EM1_low_ptr;
		EM1_high = EM1_high_ptr;
		flag = flag_ptr;
		para = _para;
	};

	__host__ __device__
	void operator()(int index) {
		int subs[3];
		int size_vec[3];
		size_vec[0] = para.nk;
		size_vec[1] = para.nz;
		size_vec[2] = para.nxxi;
		ind2sub(3,size_vec,index,subs);
		int i_k = subs[0];
		int i_z = subs[1];
		int i_xxi = subs[2];
		double k =K[i_k]; double z=Z[i_z]; double xxi=XXI[i_xxi];
		double zkttheta = z*pow(k,para.ttheta); 

		// Find the "box" or "hypercube" that described m's range. Fancy word.
		double m1min = V1_low[index]; double m1max = V1_high[index];
		double m1min_old = m1min;
		double m1max_old = m1max;
		double step1 = (m1max-m1min)/double(para.nm1-1);
		double tempflag = 0.0;
		int nm1 = para.nm1;

		// Initial search to find the min m
		for (int m_index = 0; m_index < nm1; m_index++) {
			int i_m1 = m_index;
			double m1=m1min+double(i_m1)*step1;
			if (eureka(k,z,xxi,m1,zkttheta,i_z,i_xxi,K,
				EM1_low,EM1_high,para))
			{
				tempflag++;
				m1min = m1-step1;
				break; // break and only break one layer of for loop
			};
		};

		// // "Trace-back" to refine the min_m1, assuming we found at least one m !!!
		// double min_step = step1/(nm1-1);
		// double m1min_left = m1min - step1; 
		// for (int i_m1min = 0; i_m1min < nm1; i_m1min++) {
		// 	double m1 = m1min_left + i_m1min*min_step;
		// 	if (eureka(k,z,xxi,m1,zkttheta,i_z,i_xxi,K,
		// 		EM1_low,EM1_high,para))
		// 	{
		// 		tempflag++;
		// 		m1min = m1;
		// 		break; // break and only break one layer of for loop
		// 	};
		// };

		// Initial search to find the max m
		for (int m_index = nm1-1; m_index >= 0; m_index--) {
			int i_m1 = m_index;
			double m1=m1min+double(i_m1)*step1;
			if (eureka(k,z,xxi,m1,zkttheta,i_z,i_xxi,K,
				EM1_low,EM1_high,para))
			{
				tempflag++;
				m1max = m1+step1;
				break; // break and only break one layer of for loop
			};
		};

		// // "Trace-back" to refine the max_m1, assuming we found at least one m !!!
		// double max_step = step1/(nm1-1);
		// double m1max_right = m1max + step1; 
		// for (int i_m1max = 0; i_m1max < nm1; i_m1max++) {
		// 	double m1 = m1max_right - i_m1max*max_step;
		// 	if (eureka(k,z,xxi,m1,zkttheta,i_z,i_xxi,K,
		// 		EM1_low,EM1_high,para))
		// 	{
		// 		tempflag++;
		// 		m1max = m1;
		// 		break; // break and only break one layer of for loop
		// 	};
		// };

		// Update Vs
		flag[index] = double(tempflag)/double(nm1);
		if (tempflag == 0) {
			Vplus1_high[index] = m1min_old;
			Vplus1_low[index] = m1max_old; 
		} else {
			Vplus1_high[index] = m1max;
			Vplus1_low[index] = m1min;
		};
	}
};	

// This functor calculates the error
struct myMinus {
	// Tuple is (V1low,Vplus1low,V1high,Vplus1high,...)
	template <typename Tuple>
	__host__ __device__
	double operator()(Tuple t)
	{
		return max( abs(get<0>(t)-get<1>(t)),abs(get<2>(t)-get<3>(t)) );
	}
};

// This functor calculates the distance 
struct myDist {
	// Tuple is (V1low,Vplus1low,V1high,Vplus1high,...)
	template <typename Tuple>
	__host__ __device__
	double operator()(Tuple t)
	{
		return abs(get<0>(t)-get<1>(t));
	}
};

int main()
{
	// Initialize Parameters
	para_struct para;

	// Set Accuracy Parameters
	para.nk = 256;
	para.nb = 1 ;
	para.nz = 7;
	para.nxxi = 7;
	para.nm1 = 2560 ;
	para.tol = 1e-6;
	para.maxiter = 1e5;
	para.kwidth = 1.5 ;
	para.bwidth = 1.15 ;
	para.mkwidth = 20.0 ; 

	// Set Model Parameters
	para.bbeta = 0.9825;
	para.ddelta = 0.025;
	para.ttheta = 0.36;
	para.kkappa = 0.1460;
	para.ttau = 0.3500;
	para.xxibar = 0.1;
	para.zbar = 1.0;
	para.rrhozz = 0.9457;
	para.rrhoxxiz = 0.0321;
	para.rrhozxxi =-0.0091;
	para.rrhoxxixxi = 0.9703;
	para.var_epsz = 0.0045*0.0045;
	para.var_epsxxi = 0.0098*0.0098;
	para.complete(); // complete all implied para, find S-S

	cout << setprecision(16) << "kss: " << para.kss << endl;
	cout << setprecision(16) << "zss: " << para.zbar << endl;
	cout << setprecision(16) << "xxiss: " <<para.xxibar << endl;
	cout << setprecision(16) << "mkss: " << para.mkss << endl;
	cout << setprecision(16) << "dss: " << para.dss << endl;
	cout << setprecision(16) << "css: " << para.css << endl;
	cout << setprecision(16) << "nss: " << para.nss << endl;
	cout << setprecision(16) << "wss: " << para.wss << endl;
	cout << setprecision(16) << "mmuss: " << para.mmuss << endl;
	cout << setprecision(16) << "aalpha: " << para.aalpha << endl;
	cout << setprecision(16) << "tol: " << para.tol << endl;

	// Select Device
	int num_devices;
	hipGetDeviceCount(&num_devices);
	if (num_devices>1) {
		hipSetDevice(1);
	};
	// Only for cuBLAS
	const double alpha = 1.0;
	const double beta = 0.0;

	// Create all STATE, SHOCK grids here
	host_vector<double> h_K(para.nk); 
	host_vector<double> h_Z(para.nz);
	host_vector<double> h_XXI(para.nxxi);

	host_vector<double> h_V1_low(para.nk*para.nz*para.nxxi, 1/para.mkwidth*para.mkss);
	host_vector<double> h_V1_high(para.nk*para.nz*para.nxxi,para.mkwidth*para.mkss);
	host_vector<double> h_Vplus1_low(para.nk*para.nz*para.nxxi,1/para.mkwidth*para.mkss);
	host_vector<double> h_Vplus1_high(para.nk*para.nz*para.nxxi,para.mkwidth*para.mkss);

	host_vector<double> h_EM1_low(para.nk*para.nz*para.nxxi,0.0);
	host_vector<double> h_EM1_high(para.nk*para.nz*para.nxxi,0.0);

	host_vector<double> h_P(para.nz*para.nxxi*para.nz*para.nxxi, 1.0/double(para.nz*para.nxxi));
	host_vector<double> h_flag(para.nk*para.nz*para.nxxi, 0); 

	// host_vector<double> h_c(para.nk*para.nz*para.nxxi*para.nm1);
	// host_vector<double> h_n(para.nk*para.nz*para.nxxi*para.nm1);
	// host_vector<double> h_kplus(para.nk*para.nz*para.nxxi*para.nm1);
	// host_vector<double> h_mmu(para.nk*para.nz*para.nxxi*para.nm1);
	// host_vector<double> h_lhs(para.nk*para.nz*para.nxxi*para.nm1);
	// host_vector<double> h_rhs_low(para.nk*para.nz*para.nxxi*para.nm1);
	// host_vector<double> h_rhs_high(para.nk*para.nz*para.nxxi*para.nm1);
	
	// Create capital grid
	double minK = 1/para.kwidth*para.kss;
	double maxK = para.kwidth*para.kss;
	linspace(minK,maxK,para.nk,raw_pointer_cast(h_K.data()));
	save_vec(h_K,para.nk,"Kgrid.csv");

	// Create shocks grids
	host_vector<double> h_shockgrids(2*para.nz);
	double* h_shockgrids_ptr = raw_pointer_cast(h_shockgrids.data());
	double* h_P_ptr = raw_pointer_cast(h_P.data());
	gridgen_fptr linspace_fptr = &linspace; // select linspace as grid gen
	tauchen_vec(2,para.nz,3,para.A,para.Ssigma_e,h_shockgrids_ptr,h_P_ptr,linspace_fptr);
	for (int i_shock = 0; i_shock < para.nz; i_shock++) {
		h_Z[i_shock] = para.zbar*exp(h_shockgrids[i_shock+0*para.nz]);
		h_XXI[i_shock] = para.xxibar*exp(h_shockgrids[i_shock+1*para.nz]);
	};
	display_vec(h_Z);
	display_vec(h_XXI);
	save_vec(h_Z,"Zgrid.csv");
	save_vec(h_XXI,"XXIgrid.csv");
	save_vec(h_P,"Pcuda.csv");

	// Obtain initial guess from linear solution
	guess_linear(h_K, h_Z, h_XXI, h_V1_low, h_V1_high, para, 0.5, 1.5) ;
	save_vec(h_V1_low,"V1_low_guess.csv");
	save_vec(h_V1_high,"V1_high_guess.csv");

	// Copy to the device
	device_vector<double> d_K = h_K;
	device_vector<double> d_Z = h_Z;
	device_vector<double> d_XXI = h_XXI;

	device_vector<double> d_V1_low = h_V1_low;
	device_vector<double> d_V1_high = h_V1_high;

	device_vector<double> d_Vplus1_low = h_Vplus1_low;
	device_vector<double> d_Vplus1_high = h_Vplus1_high;

	device_vector<double> d_EM1_low = h_EM1_low;
	device_vector<double> d_EM1_high = h_EM1_high;

	device_vector<double> d_P = h_P;
	device_vector<double> d_flag = h_flag;

	// Obtain device pointers to be used by cuBLAS
	double* d_K_ptr = raw_pointer_cast(d_K.data());
	double* d_Z_ptr = raw_pointer_cast(d_Z.data());
	double* d_XXI_ptr = raw_pointer_cast(d_XXI.data());

	double* d_V1_low_ptr = raw_pointer_cast(d_V1_low.data());
	double* d_V1_high_ptr = raw_pointer_cast(d_V1_high.data());

	double* d_Vplus1_low_ptr = raw_pointer_cast(d_Vplus1_low.data());
	double* d_Vplus1_high_ptr = raw_pointer_cast(d_Vplus1_high.data());

	double* d_EM1_low_ptr = raw_pointer_cast(d_EM1_low.data());
	double* d_EM1_high_ptr = raw_pointer_cast(d_EM1_high.data());

	double* d_P_ptr = raw_pointer_cast(d_P.data());
	double* d_flag_ptr = raw_pointer_cast(d_flag.data());

	// Firstly a virtual index array from 0 to nk*nk*nz
	counting_iterator<int> begin(0);
	counting_iterator<int> end(para.nk*para.nz*para.nxxi);

	// Start Timer
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,NULL);
	
	// Step.1 Has to start with this command to create a handle
	hipblasHandle_t handle;

	// Step.2 Initialize a cuBLAS context using Create function,
	// and has to be destroyed later
	hipblasCreate(&handle);
	
	double diff = 10; double dist; int iter = 0;
	while ((diff>para.tol)&&(iter<para.maxiter)){
		// Find EMs for low and high 
		hipblasDgemm(handle,
			HIPBLAS_OP_N,  
			HIPBLAS_OP_T,
			para.nk*para.nb, para.nz*para.nxxi, para.nz*para.nxxi,
			&alpha,
			d_V1_low_ptr, 
			para.nk*para.nb, 
			d_P_ptr,
			para.nz*para.nxxi,
			&beta,
			d_EM1_low_ptr,
			para.nk*para.nb);
		hipblasDgemm(handle,
			HIPBLAS_OP_N,  
			HIPBLAS_OP_T,
			para.nk*para.nb, para.nz*para.nxxi, para.nz*para.nxxi,
			&alpha,
			d_V1_high_ptr, 
			para.nk*para.nb, 
			d_P_ptr,
			para.nz*para.nxxi,
			&beta,
			d_EM1_high_ptr,
			para.nk*para.nb);

		// Directly find the new Value function
		thrust::for_each(
			begin,
			end,
			shrink(d_K_ptr, d_Z_ptr, d_XXI_ptr,
				d_V1_low_ptr,
				d_V1_high_ptr,
				d_Vplus1_low_ptr,
				d_Vplus1_high_ptr,
				d_EM1_low_ptr,
				d_EM1_high_ptr,
				d_flag_ptr,
				para)
		);

		// Find error
		double diff1 = transform_reduce(
			make_zip_iterator(make_tuple(d_V1_low.begin(), d_Vplus1_low.begin(), d_V1_high.begin(),d_Vplus1_high.begin())),
			make_zip_iterator(make_tuple(d_V1_low.end()  , d_Vplus1_low.end()  , d_V1_high.end()  ,d_Vplus1_high.end())),
			myMinus(),
			0.0,
			maximum<double>()
			);

		// Find distance 
		double dist1 = transform_reduce(
			make_zip_iterator(make_tuple(d_Vplus1_low.begin(),d_Vplus1_high.begin())),
			make_zip_iterator(make_tuple(d_Vplus1_low.end()  ,d_Vplus1_high.end())),
			myDist(),
			0.0,
			maximum<double>()
			);
		diff = max(diff1,-99.0);
		dist = max(dist1,-99.0);

		cout << "diff is: "<< diff << endl;
		cout << "dist is: "<< dist << endl;
		cout << "Vplus1[100-1] (the spike) range is " << d_Vplus1_low[100-1] << ", " << d_Vplus1_high[100-1] << endl;

		// update correspondence
		d_V1_low = d_Vplus1_low; d_V1_high = d_Vplus1_high;

		cout << ++iter << endl;
		cout << "=====================" << endl;

	};

	//==========cuBLAS stuff ends=======================
	// Step.3 Destroy the handle.
	hipblasDestroy(handle);

	// Stop Timer
	hipEventRecord(stop,NULL);
	hipEventSynchronize(stop);
	float msecTotal = 0.0;
	hipEventElapsedTime(&msecTotal, start, stop);

	// Compute and print the performance
	float msecPerMatrixMul = msecTotal;
	cout << "Time= " << msecPerMatrixMul << " msec, iter= " << iter << endl;

	// Copy back to host and print to file
	h_V1_low = d_V1_low; h_V1_high = d_V1_high;
	h_flag = d_flag;
	save_vec(h_V1_low,"V1_low.csv");
	save_vec(h_V1_high,"V1_high.csv");
	save_vec(h_flag,"flagcuda.csv");
	
	ofstream fout_kopt("koptcuda.csv", ios::trunc); ofstream fout_copt("coptcuda.csv", ios::trunc);
	ofstream fout_R("Rcuda.csv", ios::trunc);
	ofstream fout_wage("wagecuda.csv", ios::trunc);
	ofstream fout_d("dcuda.csv", ios::trunc); ofstream fout_n("ncuda.csv", ios::trunc);
	ofstream fout_Kgrid("Kgridcuda.csv", ios::trunc);
	ofstream fout_Zgrid("Zgridcuda.csv", ios::trunc); ofstream fout_XXIgrid("XXIgridcuda.csv", ios::trunc);
	ofstream fout_mmu("mmucuda.csv", ios::trunc); ofstream fout_P("Pcuda.csv", ios::trunc);
	
	double ttheta = para.ttheta;
	int nk = para.nk;
	int nz = para.nz;
	int nxxi = para.nxxi;
	for (int index=0; index<nk*nz*nxxi; index++) {
		int i_xxi = index/(nk*nz);
		int i_z  = (index-i_xxi*nk*nz)/(nk);
		int i_k = index - i_xxi*nk*nz - i_z*nk ;
		double m1 = (h_V1_low[index]+h_V1_low[index])/2;
		double k =h_K[i_k];
		double z=h_Z[i_z]; double xxi=h_XXI[i_xxi];
		double zkttheta = z*pow(k,ttheta);

		// Declare control variables
		double n, Y, MPK, kplus, c, mmu, w, d;

		// Case 1: Binding
		n = newton(case1_hour(k,z,xxi,m1,zkttheta,para),0.0,1.0,0.3);
		Y = zkttheta*pow(n,1-para.ttheta);
		MPK = para.ttheta*Y/k;
		kplus = Y/xxi;
		c = Y+(1-para.ddelta)*k-kplus;
		mmu = 1-(m1*c-1+para.ddelta)/MPK;	
		w = (1-mmu)*(1-para.ttheta)*Y/n;
		d = c - w*n;
	
		if (mmu<0)
		{
			// Case 2: Not Binding
			n = newton(case2_hour(k,z,xxi,m1,zkttheta,para),0.0,1.0,0.3);
			Y = zkttheta*pow(n,1-para.ttheta);
			MPK = para.ttheta*Y/k;
			mmu = 0;
			c = (1-para.ddelta+MPK)/m1;	
			kplus = (1-para.ddelta)*k + Y - c;
			w = (1-para.ttheta)*Y/n;
			d = c - w*n;
		};

		fout_kopt << kplus << '\n';
		fout_copt << c << '\n';
		fout_R << 1 << '\n';
		fout_d << d << '\n';
		fout_n << n << '\n';
		fout_mmu << mmu << '\n';
		fout_wage << w << '\n';
	};
	
	for (int i=0; i<nk; i++) {
		fout_Kgrid << h_K[i] << '\n';
	};
	for (int i=0; i<nz; i++) {
		fout_Zgrid << h_Z[i] << '\n';
	};
	for (int i=0; i<nxxi; i++) {
		fout_XXIgrid << h_XXI[i] << '\n';
	};
	for (int i=0; i<nz*nxxi*nz*nxxi; i++) {
		fout_P << h_P[i] << '\n';
	};

	fout_Kgrid.close();
	fout_Zgrid.close(); fout_XXIgrid.close();
	fout_kopt.close(); fout_copt.close();
	fout_R.close(); fout_d.close();
	fout_n.close(); fout_mmu.close(); fout_P.close();
	fout_wage.close();

	// Export parameters to MATLAB
	para.exportmatlab("./MATLAB/mypara.m");
	return 0;
}


